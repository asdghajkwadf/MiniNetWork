#include "hip/hip_runtime.h"
#include "CudaKernel/cudaFunc.cuh"
// #include <>
#include <hip/hip_runtime.h>
#include <stdio.h>



// 数组除一个相同datasize的数组
__global__ void Arrays_divide_arrays_Kernel(double* arr, const double n, const size_t _datasize)
{
    // 每个线程计算四个
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    for (unsigned int smallIndex = 0; smallIndex < 4; ++smallIndex)
    {   
        size_t a_index = index * 4 + smallIndex;
        if (a_index < _datasize)
        {
            arr[a_index] = arr[a_index] / n;
        }
    }
}
void __cdecl Common::Arrays_divide_arrays(double* src, const double n, size_t _datasize)
{
    const int BlockthreadNum = 512;
    const int grid = (_datasize / (512 * 4)) + 1;

    dim3 blockSize(BlockthreadNum);
    dim3 gridSize(grid);
    Arrays_divide_arrays_Kernel<<<gridSize, blockSize>>>(src, n, _datasize);
    hipDeviceSynchronize();
}





// 神经网络层误差的传递，
__global__ void AverageNextloss_kernel(double* loss, double* dst, const size_t batch_size, const size_t output_num)
{
    // 一个线程计算一个loss
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < output_num)
    {
        for (int b = 0; b < batch_size; ++b)
        {
            dst[index] += loss[b*output_num + index];
        }
        // printf("%f\n", dst[index]);
    }
}
void __cdecl FullconnecttionKernelFunc::AverageNextloss(double* loss, double* dst, const size_t batch_size, const size_t output_num)
{
    const int BlockthreadNum = 512;
    const int grid = (output_num / (512 * 4)) + 1;
    dim3 blockSize(BlockthreadNum);
    dim3 gridSize(grid);

    AverageNextloss_kernel<<<gridSize, blockSize>>>(loss, dst, batch_size, output_num);
    hipDeviceSynchronize();
}







// 更新参数所作的操作
__global__ void updata_weight_kernel(double* w, double* grad, const size_t _datasize, const double lr)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < _datasize)
    {
        w[index] -= grad[index] * lr;
        // printf("w is %f, grad is %f\n", w[index], grad[index]);
    }
}
void __cdecl Common::update_weight(double* w, double* grad, const size_t _datasize, const double lr)
{
    const int BlockthreadNum = 512;
    const int grid = (_datasize / (512)) + 1;
    dim3 blockSize(BlockthreadNum);
    dim3 gridSize(grid);
    updata_weight_kernel<<<gridSize, blockSize>>>(w, grad, _datasize, lr);
    hipDeviceSynchronize();
}






// 数组每一个元素加一个数
__global__ void batch_ouput_add_b_kernel(double* batch_output, double* _b, const size_t batch_size, const size_t output_num)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < output_num)
    {
        for (int b = 0; b < batch_size; ++b)
        {
            batch_output[b*output_num + index] += _b[index];
        }
    }
}
void FullconnecttionKernelFunc::batch_ouput_add_b(double* batch_output, double* _b, const size_t batch_size, const size_t output_num)
{
    const int BlockthreadNum = 512;
    const int grid = (output_num / (512)) + 1;
    dim3 blockSize(BlockthreadNum);
    dim3 gridSize(grid);
    batch_ouput_add_b_kernel<<<gridSize, blockSize>>>(batch_output, _b, batch_size, output_num);
    hipDeviceSynchronize();
}
