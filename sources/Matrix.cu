#include "hip/hip_runtime.h"
#include "matrix_mul.cuh"
#include <iostream>
#include <cstdlib>
#include <ctime>

__global__ void matrixMulKernel(double* A, double* B, double* result, size_t A_size, size_t B_size, size_t result_size) 
{

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // 确保线程在矩阵范围内
    if (row < A_size && col < B_size) {
        double sum = 0;
        for (int k = 0; k < K; ++k) {
            sum += A[row * K + k] * B[k * N + col];
        }
        result[row * N + col] = sum;
    }
}

// 矩阵乘法函数
void matrixMultiply(double* A, double* B, double* result, size_t A_size, size_t B_size, size_t result_size) 
{

    float *d_A, *d_B, *d_C;
    CHECK_CUDA_ERROR(hipMalloc(&d_A, M * K * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_B, K * N * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_C, M * N * sizeof(float)));

    CHECK_CUDA_ERROR(hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice));


    dim3 threadsPerBlock(16, 16); 
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrixMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, M, N, K);
    CHECK_CUDA_ERROR(hipGetLastError()); 
    CHECK_CUDA_ERROR(hipDeviceSynchronize()); 

    CHECK_CUDA_ERROR(hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));

    CHECK_CUDA_ERROR(hipFree(d_A));
    CHECK_CUDA_ERROR(hipFree(d_B));
    CHECK_CUDA_ERROR(hipFree(d_C));
}







template<temple T>
void apMem(T* dataPtr, int size)
{
    hipMalloc()
}